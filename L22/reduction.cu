
// compile with: nvcc -arch sm_60 -o reduction reduction.cu
// run: ./reduction

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

// use this later to define number of threads in thread block
#define BSIZE 256

__global__ void partialReduction(int N, 
				 float *c_a,
				 float *c_result){

  // shared memory array
  __shared__ float s_a[BSIZE];

  // find thread number in thread-block
  int t = threadIdx.x;
  
  // find block number
  int b = blockIdx.x;

  // choose an array index for this thread to read
  int n = t + b*blockDim.x;

  // check is this index in bounds
  float a = 0;
  if(n<N)
    a = c_a[n];
  
  // store the entry in shared memory
  s_a[t] = a;

  // block until all threads have written to the shared memory
  __syncthreads();
  if(t<BSIZE/2) s_a[t] = s_a[t] + s_a[t+(BSIZE/2)];

  __syncthreads();
  if(t<BSIZE/4) s_a[t] = s_a[t] + s_a[t+(BSIZE/4)];

  __syncthreads();
  if(t<BSIZE/8) s_a[t] = s_a[t] + s_a[t+(BSIZE/8)];

  __syncthreads();
  if(t<BSIZE/16) s_a[t] = s_a[t] + s_a[t+(BSIZE/16)];

  __syncthreads();
  if(t<BSIZE/32) s_a[t] = s_a[t] + s_a[t+(BSIZE/32)];

  __syncthreads();
  if(t<BSIZE/64) s_a[t] = s_a[t] + s_a[t+(BSIZE/64)];

  __syncthreads();
  if(t<BSIZE/128) s_a[t] = s_a[t] + s_a[t+(BSIZE/128)];

  __syncthreads();
  if(t<BSIZE/256) s_a[t] = s_a[t] + s_a[t+(BSIZE/256)];

  if(t==0)
    c_result[b] = s_a[0];
}


int main(int argc, char **argv){

  int N = 10240;

  // host array
  float *h_a = (float*) malloc(N*sizeof(float));
  float *h_result = (float*) malloc(N*sizeof(float));

  int n;
  for(n=0;n<N;++n){
    h_a[n] = 1;
  }

  // allocate device array
  float *c_a, *c_result;

  hipMalloc(&c_a, N*sizeof(float));
  hipMalloc(&c_result, N*sizeof(float));

  // copy data from host to device
  hipMemcpy(c_a, h_a, N*sizeof(float), hipMemcpyHostToDevice);

  // choose number of threads in thread-block
  dim3 B(BSIZE,1,1);

  // choose number of thread-blocks
  int Nblocks = (N+BSIZE-1)/BSIZE;
  int Nblocks2 = (Nblocks+BSIZE-1)/BSIZE;
  dim3 G(Nblocks,1,1);
  dim3 G2(Nblocks2,1,1);

  printf("Nblocks = %d, Nblocks2 = %d\n", Nblocks, Nblocks2);

  // launch reduction kernel
  partialReduction <<< G, B >>> (N, c_a, c_result);

  partialReduction <<< G2, B >>> (Nblocks, c_result, c_a);

  // copy result back
  hipMemcpy(h_result, c_result, 
	     Nblocks*sizeof(float), hipMemcpyDeviceToHost);

  hipMemcpy(h_result, c_a,
	     Nblocks2*sizeof(float), hipMemcpyDeviceToHost);

  // print out partial sums
  for(n=0;n<Nblocks2;++n)
    printf("%f\n", h_result[n]);

  return 0;
}
